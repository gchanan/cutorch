
#include <hip/hip_runtime.h>
/*#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCDeviceTensorUtils.cu"
#else

template <typename T, int Dim,
          typename IndexT, template <typename U> class PtrTraits>
//THC_API
THCDeviceTensor<T, Dim, IndexT, PtrTraits>
toDeviceTensor(THCState* state, THCudaTensor* t);

template <typename T, int Dim, typename IndexT>
//THC_API
THCDeviceTensor<T, Dim, IndexT, DefaultPtrTraits>
toDeviceTensor(THCState* state, THCudaTensor* t) {
  return toDeviceTensor<T, Dim, IndexT, DefaultPtrTraits>(state, t);
}

template <typename T, int Dim>
THCDeviceTensor<T, Dim, int, DefaultPtrTraits>
toDeviceTensor(THCState* state, THCudaTensor* t) {
  return toDeviceTensor<T, Dim, int, DefaultPtrTraits>(state, t);
}

#include "THCDeviceTensorUtils-inl.h"

#endif
*/
