
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCDeviceTensorUtils.cu"
#else

/// Constructs a THCDeviceTensor initialized from a THCudaTensor. Will
/// error if the dimensionality does not match exactly.
template <typename T, int Dim,
          typename IndexT, template <typename U> class PtrTraits>
THCDeviceTensor<T, Dim, IndexT, PtrTraits>
toDeviceTensor(THCState* state, THCTensor* t);

template <typename T, int Dim, typename IndexT>
THCDeviceTensor<T, Dim, IndexT, DefaultPtrTraits>
toDeviceTensor(THCState* state, THCTensor* t) {
  return toDeviceTensor<T, Dim, IndexT, DefaultPtrTraits>(state, t);
}

template <typename T, int Dim>
THCDeviceTensor<T, Dim, int, DefaultPtrTraits>
toDeviceTensor(THCState* state, THCTensor* t) {
  return toDeviceTensor<T, Dim, int, DefaultPtrTraits>(state, t);
}

/*template <typename T, int Dim,
          typename IndexT, template <typename U> class PtrTraits>
//THC_API
THCDeviceTensor<T, Dim, IndexT, PtrTraits>
toDeviceTensor(THCState* state, THCudaTensor* t);

template <typename T, int Dim, typename IndexT>
//THC_API
THCDeviceTensor<T, Dim, IndexT, DefaultPtrTraits>
toDeviceTensor(THCState* state, THCudaTensor* t) {
  return toDeviceTensor<T, Dim, IndexT, DefaultPtrTraits>(state, t);
}

template <typename T, int Dim>
THCDeviceTensor<T, Dim, int, DefaultPtrTraits>
toDeviceTensor(THCState* state, THCudaTensor* t) {
  return toDeviceTensor<T, Dim, int, DefaultPtrTraits>(state, t);
}*/

#include "THCDeviceTensorUtils-inl.cuh"

#endif
